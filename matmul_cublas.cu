#include "matmul_cublas.h"

void matcublas(complex<double> *matA, complex<double> *matB, complex<double> *matC,
            UNINT dim){

   int dim2 = dim * dim;
   // vector< complex<double> > aux_mat(dim2,0.0);
   hipDoubleComplex *dev_A, *dev_B, *dev_C;
   const hipDoubleComplex alf = make_hipDoubleComplex(1.0,0.0);
   const hipDoubleComplex bet = make_hipDoubleComplex(0.0, 0.0);
   const hipDoubleComplex *alpha = &alf;
   const hipDoubleComplex *beta = &bet;

   hipMalloc((void**) &dev_A, dim2 * sizeof(hipDoubleComplex));
   hipMalloc((void**) &dev_B, dim2 * sizeof(hipDoubleComplex));
   hipMalloc((void**) &dev_C, dim2 * sizeof(hipDoubleComplex));

   hipMemcpy(dev_A, matA, dim2 * sizeof(hipDoubleComplex),
              hipMemcpyHostToDevice);
   hipMemcpy(dev_B, matB, dim2 * sizeof(hipDoubleComplex),
              hipMemcpyHostToDevice);
   hipMemcpy(dev_C, matC, dim2 * sizeof(hipDoubleComplex),
              hipMemcpyHostToDevice);
   // Create a handle for CUBLAS
  hipblasHandle_t handle;
  hipblasCreate(&handle);

  // Do the actual multiplication

  hipblasZgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, dim, dim, dim, alpha, dev_A,
              dim, dev_B, dim, beta, dev_C, dim);

  // Destroy the handle
  hipblasDestroy(handle);

  hipMemcpy(matC, dev_C, dim2 * sizeof(hipDoubleComplex),
             hipMemcpyDeviceToHost);

   //Free GPU memory
  hipFree(dev_A);
  hipFree(dev_B);
  hipFree(dev_C);
}
